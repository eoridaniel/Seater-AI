#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <cstdlib>
#include <time.h>
#include <math.h>
#include <utility>

#define THREAD_COUNT 1024

struct Seat{
    unsigned int val : 2;
    // 0 not reserved, 1 reserved, 3 not seat 
};

class FreeSeats{
    public:
        float row;
        float first_seat;
        float free_count;

        __device__ FreeSeats(float row, float first_seat, float free_count){
            this->row = row;
            this->first_seat = first_seat;
            this->free_count = free_count;
        }
        __device__ ~FreeSeats(){
            free(&row);
            free(&first_seat);
            free(&free_count);
        }
};

class Seats{
    private:
        unsigned int max_group_size;
        unsigned int row_len;
        unsigned int col_len;
        unsigned int size;
        unsigned int fsg_size;
        FreeSeats* free_seat_groups;
        Seat* seats;        //row*row_len+col = idx
        bool* shifting;
        int* groups;

        __device__ bool is_free(int row, int first_seat, int required_seats){
            int seat_max_pos = first_seat + required_seats; 
            if(row < this->col_len && seat_max_pos <= this->row_len){
                //current row
                for(int i = (first_seat > 0 ? (first_seat - 1): first_seat) ; i < seat_max_pos; i++){
                    if(this->seats[(row*this->row_len)+i].val == (1 || 3)){
                        return false;
                    }
                }
                //previus row
                if(row > 0){
                    for(int i = ((this->shifting[row] == false && this->shifting[row-1] == false) || (this->shifting[row] && this->shifting[row-1]) ?
                    first_seat : (first_seat > 0 ? (first_seat - 1): first_seat));
                    i < (((this->shifting[row] == false && this->shifting[row-1] == false) || (this->shifting[row] && this->shifting[row-1])) ?
                    seat_max_pos  : (seat_max_pos < (row_len - 1) ? (seat_max_pos + 1) : seat_max_pos)); i++){
                        if(this->seats[((row-1)*this->row_len)+i].val == 1){
                            return false;
                        }
                    }
                }
                //next row
                if(row < (this->col_len - 1)){
                    for(int i = ((this->shifting[row] == false && this->shifting[row+1] == false) || (this->shifting[row] && this->shifting[row+1]) ?
                    first_seat : (first_seat > 0 ? (first_seat - 1): first_seat));
                    i < (((this->shifting[row] == false && this->shifting[row+1] == false) || (this->shifting[row] && this->shifting[row+1])) ?
                    seat_max_pos  : (seat_max_pos < (row_len - 1) ? (seat_max_pos + 1) : seat_max_pos)); i++){
                        if(this->seats[((row+1)*this->row_len)+i].val == 1){
                            return false;
                        }
                    }
                }
                return true;
            }
            return false;
        }
        __device__ void search_free_seat_groups(){
            int incrise = (int)((this->row_len + this->col_len) / 4);
            this->fsg_size = incrise;
            int idx = 0;
            free(this->free_seat_groups);
            this->free_seat_groups = (FreeSeats*)malloc(this->fsg_size * sizeof(FreeSeats));
            for(int row = 0; row < this->col_len; row++){
                float first_free = 0;
                float free_count = 0;
                for(int col = 0; col < this->row_len; col++){
                    if(this->is_free(row, col, 1)){
                        free_count += 1;
                        continue;
                    }else if(free_count > 0){
                        //Realloc
                        if(idx >= this->fsg_size){
                            FreeSeats* tmp = (FreeSeats*)malloc((this->fsg_size + incrise) * sizeof(FreeSeats));
                            for(int i = 0; i < this->fsg_size; i++){tmp[i] = this->free_seat_groups[i];}
                            this->fsg_size += incrise;
                            free(this->free_seat_groups);
                            this->free_seat_groups = tmp;
                        }
                        //
                        this->free_seat_groups[idx++] =  FreeSeats(row, first_free, free_count);
                    }
                    first_free = col + 1;
                    free_count = 0;
                }
                if(first_free < this->row_len){
                    //Realloc
                    if(idx >= this->fsg_size){
                        FreeSeats* tmp = (FreeSeats*)malloc((this->fsg_size + incrise) * sizeof(FreeSeats));
                        for(int i = 0; i < this->fsg_size; i++){tmp[i] = this->free_seat_groups[i];}
                        this->fsg_size += incrise;
                        free(this->free_seat_groups);
                        this->free_seat_groups = tmp;
                    }
                    //
                    this->free_seat_groups[idx++] = FreeSeats(row, first_free, free_count);
                }
            }
            //Resize the final array
            int tp = 0;
            for(int i = 0; i < this->fsg_size; i++){
                if(free_seat_groups[i].free_count < 1){tp++;}
            }
            FreeSeats* tmp = (FreeSeats*)malloc((this->fsg_size - tp) * sizeof(FreeSeats));
            for(int i = 0; i < this->fsg_size - tp; i++){tmp[i] = this->free_seat_groups[i];}
            this->fsg_size -= tp;
            free(this->free_seat_groups);
            this->free_seat_groups = tmp;
        }
        __device__ FreeSeats search_best_seats(int required_seats){
            FreeSeats best_seats = FreeSeats(INFINITY, INFINITY, INFINITY);
            if(this->fsg_size > 0){
                for(int i = 0; i < this->fsg_size; i++){
                    if(0 <= (this->free_seat_groups[i].free_count - required_seats) && (this->free_seat_groups[i].free_count - required_seats) < (best_seats.free_count - required_seats)){
                        best_seats = this->free_seat_groups[i];
                    }
                }
            }
            return best_seats;
        }
        __device__ FreeSeats search_worst_seats(int required_seats){
            FreeSeats worst_seats = FreeSeats(-INFINITY, -INFINITY, -INFINITY);
            if(this->fsg_size > 0){
                for(int i = 0; i < this->fsg_size; i++){
                    if(0 <= (this->free_seat_groups[i].free_count - required_seats) && (this->free_seat_groups[i].free_count - required_seats) > (worst_seats.free_count - required_seats)){
                        worst_seats = this->free_seat_groups[i];
                    }
                }
            }
            return worst_seats;
        }
        __device__ void best_fit(int required_seats){
            if(required_seats <= this->max_group_size){
                this->search_free_seat_groups();
                FreeSeats best_seats = this->search_best_seats(required_seats);
                if(best_seats.free_count != INFINITY && this->is_free(best_seats.row, best_seats.first_seat, required_seats)){
                    this->reserve(best_seats.row, best_seats.first_seat, required_seats);
                    return;
                }
            }
        }
        __device__ void worst_fit(int required_seats){
            if(required_seats <= this->max_group_size){
                this->search_free_seat_groups();
                FreeSeats worst_seats = this->search_worst_seats(required_seats);
                if(worst_seats.free_count != -INFINITY && this->is_free(worst_seats.row, worst_seats.first_seat, required_seats)){
                    this->reserve(worst_seats.row, worst_seats.first_seat, required_seats);
                    return;
                }
            }
        }
        __device__ void first_fit(int required_seats){
            if(this->max_group_size >=  required_seats){
                for(int i = 0; i <  this->size; i++){
                    if(this->is_free((int)(i/this->row_len),i % this->row_len, required_seats)){
                        this->reserve((int)(i/this->row_len), i % this->row_len, required_seats);
                        return;
                    }
                }
            }
        }
        __device__ void reserve(int row, int first_seat, int required_seats){
            for(int i = (row * this->row_len + first_seat); i < (row * this->row_len + first_seat + required_seats); i++){
                this->seats[i].val = 1;
            }
        }
        __device__ int score(){
            int reserved_count = 0;
            for(int i = 0; i < this->size; i++){
                if(this->seats[i].val == 1){
                    reserved_count++;
                }
            }
            return reserved_count;
        }

    public:
        __device__ Seats(int max_group_size, Seat *seats, bool *shifting, int row_len, int col_len, int* groups){
            this->max_group_size = max_group_size;
            this->row_len = row_len;
            this->col_len = col_len;
            this->groups = groups;
            this->seats = seats;
            this->shifting = shifting;
            this->size = this->row_len * this->col_len;
        }
        ~Seats(){
            free(&this->row_len);
            free(&this->col_len);
            free(&this->max_group_size);
            free(&this->size);
        }
        __device__ int wf_simulate(){
            for(int i = 0; i < (int)(this->size * 0.6); i++){
                this->worst_fit(this->groups[i]);
            }
            return this->score();
        }
        __device__ int bf_simulate(){
            for(int i = 0; i < (int)(this->size * 0.6); i++){
                this->best_fit(this->groups[i]);
            }
            return this->score();
        }
        __device__ int ff_simulate(){
            for(int i = 0; i < (int)(this->size * 0.6); i++){
                this->first_fit(this->groups[i]);
            }
            return this->score();
        }
};

__global__ void train(Seat** seats, bool* shifting, int max_group_size, int row_len, int col_len, int* res, int** groups, int block_idx, int block_count){
    Seats *a = new Seats(max_group_size, seats[(block_idx * THREAD_COUNT) + threadIdx.x], shifting, row_len,col_len, groups[(block_idx * THREAD_COUNT) + threadIdx.x]);
    if((block_idx * THREAD_COUNT) + threadIdx.x > (block_count * THREAD_COUNT / 3 * 2)){
        res[(block_idx * THREAD_COUNT) + threadIdx.x] = a->wf_simulate();
    }else if((block_idx * THREAD_COUNT) + threadIdx.x > (block_count * THREAD_COUNT / 3)){
        res[(block_idx * THREAD_COUNT) + threadIdx.x] = a->bf_simulate();
    }
    else{
        res[(block_idx * THREAD_COUNT) + threadIdx.x] = a->ff_simulate();
    }
    printf("%i,", res[block_idx * THREAD_COUNT + threadIdx.x]);
} 

int main(int argc, char * argv[]){

    //Set simulation parameters
    clock_t start = clock();
    unsigned int sim_count = 10000;
    const unsigned int block_count = ((int)sim_count % THREAD_COUNT == 0) ? (sim_count/THREAD_COUNT) : ((int)(sim_count/THREAD_COUNT+1));
    sim_count = block_count * THREAD_COUNT;

    //Prepare rooms and rooms' parameters
    unsigned int col_len  = 35, row_len = 35, max_group_size = 4, group_count = ((int)(row_len * col_len * 0.6));
    
    srand(time(NULL));
    int** groups = (int**)malloc(sim_count*sizeof(int*));
    for(int i = 0; i < sim_count; i++){
        groups[i] = (int*)malloc(group_count*sizeof(int));
        for(int j = 0; j < group_count; j++){
            groups[i][j] = rand() % max_group_size + 1;
        }
    }
    
    int* res = (int*)malloc(sim_count*sizeof(int));
    for(int i = 0; i < sim_count; i++){
        res[i] = 0;
    }

    Seat**  seats = (Seat**)malloc(sim_count*sizeof(Seat*));
    for (int i = 0; i < sim_count; i++){
        seats[i] = (Seat*) malloc(row_len*col_len*sizeof(Seat));
        for(int j = 0; j < row_len*col_len; j++){
            seats[i][j].val = 0;
        }
    }

    bool* shifting = (bool*)malloc(col_len*sizeof(bool));
    for(int i = 0 ; i < col_len; i++){
        shifting[i] = false;
    }

    //Allocate memory on GPU and copy data into VRAM
    Seat** d_seats;
    Seat* temp[sim_count];
    bool* d_shifting;
    int* d_res;
    int** d_groups;
    int* gtemp[sim_count];
    

    hipMalloc((void**)&d_seats, sim_count*sizeof(Seat*));
    for(int i = 0; i < sim_count; i++){
        hipMalloc(&(temp[i]), row_len*col_len*sizeof(Seat));
    }
    hipMemcpy(d_seats,temp,sizeof(Seat*)*sim_count,hipMemcpyHostToDevice);
    for(int i = 0; i < sim_count; i++){
        hipMemcpy(temp[i], seats[i], row_len*col_len*sizeof(Seat), hipMemcpyHostToDevice);
    }
    hipMalloc((void**)&d_shifting,sizeof(bool)*col_len);
    hipMemcpy(d_shifting,shifting,sizeof(bool)*col_len,hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_res,sizeof(int)*sim_count);
    hipMemcpy(d_res, res, sizeof(int)*sim_count, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_groups, sim_count*sizeof(int*));
    for(int i = 0; i < sim_count; i++){
        hipMalloc(&(gtemp[i]), group_count*sizeof(int));
    }
    hipMemcpy(d_groups,gtemp,sizeof(int*)*sim_count,hipMemcpyHostToDevice);
    for(int i = 0; i < sim_count; i++){
        hipMemcpy(gtemp[i], groups[i], group_count*sizeof(Seat), hipMemcpyHostToDevice);
    }

    //Simulate
    for(int i = 0; i < block_count; i++){    
        printf("%i\n", i*THREAD_COUNT);    
        train<<<1, THREAD_COUNT>>>(d_seats, d_shifting, max_group_size, row_len, col_len, d_res, d_groups, i, block_count);
        /*for(int j = (i*THREAD_COUNT); j < ((i+1)*THREAD_COUNT); j++){
            cudaMemcpy(seats[j], temp[j], sizeof(Seat)*row_len*col_len, cudaMemcpyDeviceToHost);
        }
        cudaMemcpy(res, d_res, sizeof(int)*sim_count, cudaMemcpyDeviceToHost);*/
    }
    
    for(int i = 0; i < sim_count; i++){
        printf("%i %i,", res[i], seats[i][0].val);
    }

    for(int i = 0; i < sim_count; i++){
        hipFree(temp[i]);
    }
    hipFree(d_seats);
    for(int i = 0; i < sim_count; i++){
        hipFree(gtemp[i]);
    }
    hipFree(d_groups);
    hipFree(d_res);
    hipFree(d_shifting);
    clock_t stop = clock();
    printf("\n%f ", ((double)((stop-start)/CLOCKS_PER_SEC)));

    //Write Rresoult into database

    return 0;
}