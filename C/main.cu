#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define BLOCK_SIZE 256

__global__ void vector_add(float *a, float *b ,float *out, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x* gridDim.x;
    for (int i = index; i < n; i += stride){
        out[i] = a[i]+ b[i];
    }
}

int main(int argc, char * argv[]){
    
    float *h_a, *h_b, *h_out;

    float *d_a, *d_b, *d_out;

    h_a = (float*)malloc(sizeof(float)* N);
    h_b = (float*)malloc(sizeof(float)* N);
    h_out = (float*)malloc(sizeof(float)* N);

    for(int i = 0;i<N;i++)
    {
        h_a[i] = 0.1f;
        h_b[i] = 0.2f;
        h_out[i] = 0;
    }

    hipMalloc((void**)&d_a,sizeof(float)*N);
    hipMemcpy(d_a,h_a,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_b,sizeof(float)*N);
    hipMemcpy(d_b,h_b,sizeof(float)*N,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_out,sizeof(float)*N);
    hipMemcpy(d_out,h_out,sizeof(float)*N,hipMemcpyHostToDevice);

    int BLOCKS_NUM = (N+BLOCK_SIZE-1)/BLOCK_SIZE;
    vector_add<<<BLOCKS_NUM,BLOCK_SIZE>>>(d_a,d_b, d_out,N);

    hipDeviceSynchronize();
    hipMemcpy(h_out,d_out,sizeof(float)*N, hipMemcpyDeviceToHost);

    for(int i = 0; i<N; i++)
    {
        printf("%f\n",h_out[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(h_a);
    free(h_b);
    free(h_out);



    return 0;
}
